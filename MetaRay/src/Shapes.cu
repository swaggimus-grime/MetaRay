#include "hip/hip_runtime.h"
#include "Shapes.h"

Sphere::Sphere(const point3& center, float radius, Material* mat)
	:m_Center(center), m_Radius(radius), m_Mat(mat)
{
}

bool Sphere::hit(const ray& r, float t_min, float t_max, hit_record& rec) const
{
    vec3 oc = r.origin - m_Center;
    float a = r.direction.dot(r.direction);
    float b = oc.dot(r.direction);
    float c = oc.dot(oc) - m_Radius * m_Radius;
    float discriminant = b * b - a * c;
    if (discriminant > 0) {
        float temp = (-b - sqrt(discriminant)) / a;
        if (temp < t_max && temp > t_min) {
            rec.t = temp;
            rec.p = r.at(rec.t);
            rec.normal = (rec.p - m_Center) / m_Radius;
            rec.mat = m_Mat;
            return true;
        }
        temp = (-b + sqrt(discriminant)) / a;
        if (temp < t_max && temp > t_min) {
            rec.t = temp;
            rec.p = r.at(rec.t);
            rec.normal = (rec.p - m_Center) / m_Radius;
            rec.mat = m_Mat;
            return true;
        }
    }
    return false;
}

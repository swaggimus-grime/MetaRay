#include "hip/hip_runtime.h"
#include "Material.h"
#include "Material.h"

Diffuse::Diffuse(const color& a)
    : m_Albedo(a)
{
}

bool Diffuse::Scatter(const ray& r_in, const hit_record& rec,
    color& attenuation, ray& scattered, hiprandState* local_rand_state) const
{
    vec3 scatter_direction = rec.normal + vec3::random_unit(local_rand_state);

    if (scatter_direction.near_zero())
        scatter_direction = rec.normal;

    scattered = ray{ rec.p, scatter_direction };
    attenuation = m_Albedo;
    return true;
}

Metal::Metal(const color& a, float fuzz)
: m_Albedo(a), m_Fuzz(fuzz < 1 ? fuzz : 1) 
{
}

bool Metal::Scatter(const ray & r_in, const hit_record & rec,
    color& attenuation, ray& scattered, hiprandState* local_rand_state) const
{
    vec3 reflected = r_in.direction.unit().reflect(rec.normal);
    scattered = ray{ rec.p, reflected + vec3::random_in_unit_sphere(local_rand_state) * m_Fuzz };
    attenuation = m_Albedo;
    return scattered.direction.dot(rec.normal) > 0;
}

Dielectric::Dielectric(float index_of_refraction)
: ir(index_of_refraction) 
{
}

bool Dielectric::Scatter(const ray& r_in, const hit_record& rec, 
    color& attenuation, ray& scattered, hiprandState* local_rand_state) const
{
    attenuation = color(1.0, 1.0, 1.0);
    float refraction_ratio = rec.front_face ? (1.0 / ir) : ir;

    vec3 unit_direction = r_in.direction.unit();
    double cos_theta = -unit_direction.dot(rec.normal);
    cos_theta = cos_theta < 1.0 ? cos_theta : 1.0;
    double sin_theta = sqrt(1.0 - cos_theta * cos_theta);

    bool cannot_refract = refraction_ratio * sin_theta > 1.0;
    vec3 direction;

    if (cannot_refract || Reflectance(cos_theta, refraction_ratio) > Random::Float(local_rand_state))
        direction = unit_direction.reflect(rec.normal);
    else
        direction = unit_direction.refract(rec.normal, refraction_ratio);

    scattered = ray{ rec.p, direction };
    return true;
}

float Dielectric::Reflectance(float cosine, float ref_idx) const
{
    // Use Schlick's approximation for reflectance.
    auto r0 = (1 - ref_idx) / (1 + ref_idx);
    r0 = r0 * r0;
    return r0 + (1 - r0) * pow((1 - cosine), 5);
}

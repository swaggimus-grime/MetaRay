#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include "vec3.h"
#include "ray.h"
#include "Shapes.h"
#include "Hittable.h"
#include "Camera.h"
#include "Material.h"
#include <fstream>
#include <thread>
#include <future>
#include <sstream>
#include <queue>

__device__ vec3 ray_color(const ray& r, Hittable** world, hiprandState* local_rand_state) {
    ray cur_ray = r;
    vec3 cur_attenuation = vec3(1.0, 1.0, 1.0);
    for (int i = 0; i < 50; i++) {
        hit_record rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            vec3 attenuation;
            if (rec.mat->Scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return vec3(0.0, 0.0, 0.0);
            }
        }
        else {
            vec3 unit_direction = cur_ray.direction.unit();
            float t = 0.5f * (unit_direction.y + 1.0f);
            vec3 c =  vec3(1.0, 1.0, 1.0) * (1.0f - t) + vec3(0.5, 0.7, 1.0) * t;
            return cur_attenuation * c;
        }
    }
    return vec3(0.0, 0.0, 0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3* fb, int max_x, int max_y, int ns, Camera** cam, Hittable** world, hiprandState* rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j * max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    vec3 col(0, 0, 0);
    for (int s = 0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->LookAt(u, v, &local_rand_state);
        col += ray_color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))

__global__ void create_world(Hittable** d_list, Hittable** d_world, Camera** d_camera, int nx, int ny, hiprandState* rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;
        d_list[0] = new Sphere(vec3(0, -1000.0, -1), 1000,
            new Diffuse(vec3(0.5, 0.5, 0.5)));
        int i = 1;
        for (int a = -11; a < 11; a++) {
            for (int b = -11; b < 11; b++) {
                float choose_mat = RND;
                vec3 center(a + RND, 0.2, b + RND);
                if (choose_mat < 0.8f) {
                    d_list[i++] = new Sphere(center, 0.2,
                        new Diffuse(vec3(RND * RND, RND * RND, RND * RND)));
                }
                else if (choose_mat < 0.95f) {
                    d_list[i++] = new Sphere(center, 0.2,
                        new Metal(vec3(0.5f * (1.0f + RND), 0.5f * (1.0f + RND), 0.5f * (1.0f + RND)), 0.5f * RND));
                }
                else {
                    d_list[i++] = new Sphere(center, 0.2, new Dielectric(1.5));
                }
            }
        }
        d_list[i++] = new Sphere(vec3(0, 1, 0), 1.0, new Dielectric(1.5));
        d_list[i++] = new Sphere(vec3(-4, 1, 0), 1.0, new Diffuse(vec3(0.4, 0.2, 0.1)));
        d_list[i++] = new Sphere(vec3(4, 1, 0), 1.0, new Metal(vec3(0.7, 0.6, 0.5), 0.0));
        *rand_state = local_rand_state;
        *d_world = new HitList(d_list, 22 * 22 + 1 + 3);

        vec3 lookfrom(13, 2, 3);
        vec3 lookat(0, 0, 0);
        float dist_to_focus = 10.0; (lookfrom - lookat).len();
        float aperture = 0.1;
        *d_camera = new Camera(lookfrom,
            lookat,
            vec3(0, 1, 0),
            30.0,
            float(nx) / float(ny),
            aperture,
            dist_to_focus);
    }
}

__global__ void free_world(Hittable** d_list, Hittable** d_world, Camera** d_camera) {
    for (int i = 0; i < 22 * 22 + 1 + 3; i++) {
        delete ((Sphere*)d_list[i])->m_Mat;
        delete d_list[i];
    }
    delete* d_world;
    delete* d_camera;
}

int main() {
    int nx = 1920;
    int ny = 1080;
    int ns = 100;
    int tx = 16;
    int ty = 16;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    size_t fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    vec3* fb;
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // allocate random state
    hiprandState* d_rand_state;
    checkCudaErrors(hipMalloc((void**)&d_rand_state, num_pixels * sizeof(hiprandState)));
    hiprandState* d_rand_state2;
    checkCudaErrors(hipMalloc((void**)&d_rand_state2, 1 * sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init << <1, 1 >> > (d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the camera
    Hittable** d_list;
    int num_hitables = 22 * 22 + 1 + 3;
    checkCudaErrors(hipMalloc((void**)&d_list, num_hitables * sizeof(Hittable*)));
    Hittable** d_world;
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(Hittable*)));
    Camera** d_camera;
    checkCudaErrors(hipMalloc((void**)&d_camera, sizeof(Camera*)));
    create_world << <1, 1 >> > (d_list, d_world, d_camera, nx, ny, d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    render_init << <blocks, threads >> > (nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render << <blocks, threads >> > (fb, nx, ny, ns, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::fstream img;
    img.open("image.ppm");
    img << "P3\n" << nx << " " << ny << "\n255\n";

    auto to_string = [&nx, &fb](int beg, int end) {
        std::stringstream ss;

        for (; end >= beg; end--)
            for (int i = 0; i < nx; i++) {
                size_t pixel_index = end * nx + i;
                int ir = int(255.99 * fb[pixel_index].r);
                int ig = int(255.99 * fb[pixel_index].g);
                int ib = int(255.99 * fb[pixel_index].b);
                ss << ir << " " << ig << " " << ib << "\n";
            }

        return ss;
    };

    int num_workers = 4;
    float linesPerWorker = ny / num_workers;
    float leftoverLines = ny % num_workers;
    float end = std::roundf(linesPerWorker);
    std::stringstream ss;
    int begIdx = linesPerWorker * (num_workers - 1);
    float endIdx = begIdx + linesPerWorker - 1 + leftoverLines;
    std::queue<std::future<std::stringstream>> workers;
    for (int i = 0; i < num_workers; i++) {
        workers.push(std::async(to_string, begIdx, endIdx));
        endIdx = begIdx - 1;
        begIdx -= linesPerWorker;
    }

    for (int i = 0; i < num_workers; i++) {
        auto& w = workers.front();
        w.wait();
        img << w.get().str();
        workers.pop();
    }
    img.close();

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world << <1, 1 >> > (d_list, d_world, d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}